#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include "sha1.cu"

#define HASH_SIZE 20
#define MESSAGE_SIZE 2

__global__
void cudaHash(
	unsigned char* hash,
	const unsigned char* message,
	int length
)
{	
	printf("in kernel\n");
	SHA1(hash,message,length);
	printf("Message: %s\n",message);
	printf("Hash: ");
	for(int i=0;i<20;i++)
		printf("%02x",hash[i]);
	printf("\n");
	printf("leaving kernel\n");
}



int main(){
	//computing the hash with verified libraries for error checking
	const unsigned char *message = (const unsigned char *)"aa";
	unsigned char hash[HASH_SIZE];

	unsigned char* d_message,*d_hash;
	hipMalloc( &d_message,MESSAGE_SIZE*sizeof(unsigned char) );
	hipMalloc( &d_hash,HASH_SIZE*sizeof(unsigned char) );
	hipMemcpy(d_message,message,MESSAGE_SIZE*sizeof(unsigned char),
		hipMemcpyHostToDevice);

	cudaHash<<<1,1>>>(d_hash,d_message,MESSAGE_SIZE);

	hipMemcpy(hash,d_hash,HASH_SIZE*sizeof(unsigned char),
		hipMemcpyDeviceToHost);
	hipFree(d_hash);
	hipFree(d_message);

	//SHA1(hash,message, MESSAGE_SIZE);
	printf("Message: %s\n",message);
	printf("Hash: ");
	for(int i=0;i<20;i++)
		printf("%02x",hash[i]);
	printf("\n");


	return 0;
}